#include "hip/hip_runtime.h"
/*
 *  dmm_gpu.cu -- Template for DMM GPU kernels
 *
 *  Copyright (C) 2019, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2019, Athena Elafrou
 */

#include "dmm.h"
#include <hipblas.h>

/*
 *  Naive kernel
 */
__global__ void dmm_gpu_naive(const value_t *A, const value_t *B, value_t *C,
                              const size_t M, const size_t N, const size_t K) {
    /*
   * FILLME: fill the code.
   */
    for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < M; i += blockDim.y * gridDim.y)
        for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < N; j += blockDim.x * gridDim.x) {
            value_t _Cij = 0;

            for (int k = 0; k < K; ++k)
                _Cij += A[i * K + k] * B[k * N + j];

            C[i * N + j] = _Cij;
        }
}

/*
 *  Coalesced memory acceses of A.
 */
__global__ void dmm_gpu_coalesced_A(const value_t *A, const value_t *B,
                                    value_t *C, const size_t M, const size_t N,
                                    const size_t K) {
    /*
   * FILLME: fill the code.
   */
    for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < M; i += blockDim.y * gridDim.y)
        for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < N; j += blockDim.x * gridDim.x) {
            value_t _Cij = 0;

            for (int m = 0; m < K; m += TILE_X) {
                __shared__ value_t As[THREAD_BLOCK_Y][TILE_X];

                for (int n = threadIdx.x; n < TILE_X; n += blockDim.x)
                    As[threadIdx.y][n] = A[i * K + n + m];

                __syncthreads();

                for (int k = 0; k < TILE_X; ++k)
                    _Cij += As[threadIdx.y][k] * B[(k + m) * N + j];

                __syncthreads();
            }

            C[i * N + j] = _Cij;
        }
}

/*
 *  Reduced memory accesses.
 */
__global__ void dmm_gpu_reduced_global(const value_t *A, const value_t *B, value_t *C,
                                       const size_t M, const size_t N, const size_t K) {
    /*
   * FILLME: fill the code.
   */
    for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < M; i += blockDim.y * gridDim.y)
        for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < N; j += blockDim.x * gridDim.x) {
            value_t _Cij = 0;

            for (int m = 0; m < K; m += TILE_X) {
                __shared__ value_t As[THREAD_BLOCK_Y][TILE_X];
                __shared__ value_t Bs[TILE_X][THREAD_BLOCK_X];

                for (int n = threadIdx.x; n < TILE_X; n += blockDim.x)
                    As[threadIdx.y][n] = A[i * K + n + m];
                for (int n = threadIdx.y; n < TILE_X; n += blockDim.y)
                    Bs[n][threadIdx.x] = B[(n + m) * N + j];

                __syncthreads();

                for (int k = 0; k < TILE_X; ++k)
                    _Cij += As[threadIdx.y][k] * Bs[k][threadIdx.x];

                __syncthreads();
            }

            C[i * N + j] = _Cij;
        }
}

/*
 *  Use of cuBLAS
 */
void dmm_gpu_cublas(const value_t *A, const value_t *B, value_t *C,
                    const size_t M, const size_t N, const size_t K) {
    /*
   * FILLME: fill the code.
   */
    const float alpha = 1, beta = 0;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, A, M, B, K, &beta, C, M);
    hipblasDestroy(handle);
}
